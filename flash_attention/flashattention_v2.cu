#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>

constexpr int WARP_SIZE = 32;
constexpr int d = 64;

__host__ __device__ inline constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }

template<int BLOCK_SIZE, int Br, int Bc> 
__global__ void flashattn_kernel_v2(
    const float *Q, //  [B, nh, T, head_dim]
    const float *K, //  [B, nh, T, head_dim]
    const float *V, //  [B, nh, T, head_dim]
    float *__restrict__ O, //  [B, nh, T, head_dim]
    float *l, // norm intermediate [B, nh, T]
    float *m, // Row max intermediate [B, nh, T]
    int B, int nh, int T, 
    float scale // Scale factor (usually 1/sqrt(head_dim))
) { 
    int tid = threadIdx.x;
    int batch_head_id = blockIdx.x;

    int s_row = tid / Bc;
    int s_col = tid % Bc;
    
    // Register variables for this thread's row statistics
    float my_m_i = -INFINITY;  // Each thread maintains its own m_i
    float my_l_i = 0.0f;       // Each thread maintains its own l_i

    const int Tc = cdiv(T, Bc); const int Tr = cdiv(T, Br);

    const int QKV_head_offset = batch_head_id * T * d;
    const int lm_head_offset = batch_head_id * T;

    int num_tiles_kv = cdiv(d, Br);
    int num_tiles_q = cdiv(d, Bc);

    __shared__ float Q_smem[Br * d];
    __shared__ float K_smem[Bc * d];
    __shared__ float V_smem[Bc * d];
    __shared__ float S_ij_smem[Br * Bc];
    __shared__ float O_smem[Br * d];
    __shared__ float m_new[Br];
    __shared__ float l_new[Br];
    __shared__ float m_ij_smem[Br];

    for (int j = 0; j < Tc; ++j) {
        // Load K and V tiles into shared memory
        for (int x = 0; x < num_tiles_kv; ++x) {
            int idx = x * (Bc * Br) + tid;
            if (idx < Bc * d) {
                int kv_row = idx / d;
                int kv_col = idx % d;
                if (j * Bc + kv_row < T) {
                    K_smem[kv_row * d + kv_col] = K[QKV_head_offset + (j * Bc + kv_row) * d + kv_col];
                    V_smem[kv_row * d + kv_col] = V[QKV_head_offset + (j * Bc + kv_row) * d + kv_col];
                }
            }
        }
        
        __syncthreads();

        // For causal attention, we only process blocks where i >= j
        for (int i = j; i < Tr; ++i) {
            // Load Q and O tiles into shared memory
            for (int x = 0; x < num_tiles_q; ++x) {
                int idx = x * (Br * Bc) + tid;
                if (idx < Br * d) {
                    int qo_row = idx / d;
                    int qo_col = idx % d; 
                    if (i * Br + qo_row < T) {
                        Q_smem[qo_row * d + qo_col] = Q[QKV_head_offset + (i * Br + qo_row) * d + qo_col];
                        O_smem[qo_row * d + qo_col] = O[QKV_head_offset + (i * Br + qo_row) * d + qo_col];
                    }
                }
            }
            __syncthreads();
            
            // Load m_i and l_i from global memory into registers for this thread's row
            int global_row = i * Br + s_row;
            if (global_row < T) {
                my_m_i = m[lm_head_offset + global_row];
                my_l_i = l[lm_head_offset + global_row];
            }

            // Compute S_ij = Q_i * K_j^T
            float acc = 0.0f;
            for (int k = 0; k < d; k++) {
                acc += Q_smem[s_row * d + k] * K_smem[s_col * d + k];
            }
            acc *= scale;

            // Apply causal masking
            int query_pos = i * Br + s_row;
            int key_pos = j * Bc + s_col;
            if (key_pos > query_pos) {
                acc = -INFINITY;
            }
            
            S_ij_smem[s_row * Bc + s_col] = acc;
            __syncthreads();

            // Compute row max and softmax - distributed across threads
            // Each thread handles one element in the row
            float my_val = S_ij_smem[s_row * Bc + s_col];
            
            // Find row maximum using warp shuffle reduction
            float m_ij = my_val;
            for (int offset = Bc/2; offset > 0; offset /= 2) {
                float other_val = __shfl_down_sync(0xFFFFFFFF, m_ij, offset);
                m_ij = fmaxf(m_ij, other_val);
            }
            // Broadcast the max back to all threads in the row
            m_ij = __shfl_sync(0xFFFFFFFF, m_ij, 0);
            
            // Compute exponential and sum
            float exp_val = __expf(my_val - m_ij);
            S_ij_smem[s_row * Bc + s_col] = exp_val;
            
            // Sum reduction using warp shuffle
            float l_ij = exp_val;
            for (int offset = Bc/2; offset > 0; offset /= 2) {
                l_ij += __shfl_down_sync(0xFFFFFFFF, l_ij, offset);
            }
            // Broadcast the sum back to all threads
            l_ij = __shfl_sync(0xFFFFFFFF, l_ij, 0);
            
            // Only one thread per row updates the shared arrays
            if (s_col == 0) {
                m_ij_smem[s_row] = m_ij;
                m_new[s_row] = fmaxf(m_ij, my_m_i);
                l_new[s_row] = __expf(my_m_i - m_new[s_row]) * my_l_i + __expf(m_ij - m_new[s_row]) * l_ij;
            }
            __syncthreads();

            float alpha = __expf(my_m_i - m_new[s_row]);  // Use register value
            float beta = __expf(m_ij_smem[s_row] - m_new[s_row]);
            
            // Compute S_ij * V_j for this thread's column
            for (int col = s_col; col < d; col += Bc) {
                float PV_acc = 0.0f;
                for (int k = 0; k < Bc; k++) {
                    PV_acc += S_ij_smem[s_row * Bc + k] * V_smem[k * d + col];
                }
                
                // Update output with the running computation
                float o_old = O_smem[s_row * d + col];
                float o_new = (1.0f / l_new[s_row]) * (my_l_i * alpha * o_old + beta * PV_acc);  // Use register value
                O_smem[s_row * d + col] = o_new;
                O[QKV_head_offset + (i * Br + s_row) * d + col] = o_new;
            }
            
            // Update register values for next iteration
            my_m_i = m_new[s_row];
            my_l_i = l_new[s_row];
            
            // Write back to global memory
            if (global_row < T) {
                m[lm_head_offset + global_row] = my_m_i;
                l[lm_head_offset + global_row] = my_l_i;
            }
            __syncthreads();
        }
        __syncthreads();
    }
} 

// Host LAUNCHER function
void flashattn_v2(const float *Q, const float *K, const float *V, float *O,
                  float *l, float *m, 
                  int B, int nh, int T, int d) {
    const int Bc = 32; const int Br = 32;
    const int BLOCK_SIZE = Bc * Br;
    const float scale = 1.0 / sqrt(d);    

    dim3 grid_dim(B * nh);  // batch_size x num_heads
    dim3 block_dim(BLOCK_SIZE);

    flashattn_kernel_v2<BLOCK_SIZE, Br, Bc><<<grid_dim, block_dim>>>(
        Q, K, V, O, l, m, B, nh, T, scale
    );
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error after kernel launch: %s\n", hipGetErrorString(err));
    }
}