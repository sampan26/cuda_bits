#include "hip/hip_runtime.h"
#include "ptx.cuh"

namespace M7 {

typedef __hip_bfloat16 bf16;

template <int BlockMajorSize, int BlockMinorSize>
__host__ static inline CUtensorMap create_tensor_map(bf16* gmem_ptr, int global_height, int global_width) {
    CUtensorMap tma_map;
    void* gmem_address = (void*)gmem_ptr;
    static_assert(BlockMinorSize >= 64);
    assert(global_width % 64 == 0);
    uint64_t gmem_prob_shape[5] = {64, (uint64_t)global_height, (uint64_t)global_width/64, 1, 1};
    uint64_t gmem_prob_stride[5] = {sizeof(bf16) * global_width, 64*sizeof(bf16), 0, 0, 0};
    uint32_t smem_box_shape[5] = {64, uint32_t(BlockMajorSize), uint32_t(BlockMinorSize/64), 1, 1};
    uint32_t smem_box_stride[5] = {1, 1, 1, 1, 1};

    hipError_t result = cuTensorMapEncodeTiled(
        &tma_map, CU_TENSOR_MAP_DATA_TYPE_BFLOAT16, 3, gmem_address, gmem_prob_shape,
        gmem_prob_stride, smem_box_shape, smem_box_stride, CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_128B, CU_TENSOR_MAP_L2_PROMOTION_NONE, CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

    assert(result == hipSuccess);
    return tma_map;
}

CUtensorMap d_tma_map_A;
CUtensorMap d_tma_map_B;

template <uint32_t RegCount>
__device__ void warpgroup_reg_alloc() {
        asm volatile("setmaxnreg.inc.sync.aligned.u32 %0;\n" : : "n"(RegCount));
}

template <uint32_t RegCount>
__device__ void warpgroup_reg_dealloc() {
        asm volatile("setmaxnreg.dec.sync.aligned.u32 %0;\n" : : "n"(RegCount));
}

template <int BM, int BN, int BK, int PIPE>
struct SharedStorage {
    alignas(128) bf16 A[BM*BK*PIPE];
    alignas(128) bf16 B[BK*BN*PIPE];
};

__device__ void calculate_tile_indices(int tile_idx, int num_blocks_n, int group_size_m, int group_size_n, int tiles_in_group, int& tile_m, int& tile_n) {
    int group_idx = tile_idx / tiles_in_group;
    int tile_idx_in_group = tile_idx % tiles_in_group;
    int group_m = group_idx / (num_blocks_n / group_size_n);
    int group_n = group_idx % (num_blocks_n / group_size_n);
    int tile_group_m = tile_idx_in_group / group_size_n;
    int tile_group_n = tile_idx_in_group % group_size_n;
    tile_m = group_m * group_size_m + tile_group_m;
    tile_n = group_n * group_size_n + tile_group_n;
}

template<int BM, int BN, int BK, int NUM_THREADS, int PIPE, int NUM_SM, int CLUSTER_M, int CLUSTER_N>
__global__  __launch_bounds__(NUM_THREADS) 
void __cluster_dim__(CLUSTER_M * CLUSTER_N, 1, 1)
matmul_kernel_v7(int M, int N, int K, bf16* C, const __grid_constant__ CUtensorMap tensorMapA, const __grid_constant__ CUtensorMap tensorMapB) {
    constexpr int WGMMA_M = 64, WGMMA_K = 16, WGMMA_N = BN;
    constexpr int num_consumers = (NUM_THREADS / 128) - 1;
    constexpr int B_WG_M = BM / num_consumers;
    constexpr int CLUSTERS = CLUSTER_M * CLUSTER_N;

    extern __shared__ __align__(128) uint8_t smem[];
    SharedStorage<BM, BN, BK, PIPE> &s = *reinterpret_cast<SharedStorage<BM, BN, BK, PIPE>*>(smem);
    bf16 *sA = s.A;
    bf16 *sB = s.B;

    __shared__ __align__(8) uint64_t full_barrier[PIPE], empty_barrier[PIPE];
    uint32_t cluster_id, rank;
    asm volatile("mov.u32 %0, %clusterid.x;\n" : "r"(cluster_id) :);

    const int num_tiles_k = K / BK;
    const int num_blocks_m = M / (BM * CLUSTER_M);
    const int num_blocks_n = N / (BN * CLUSTER_N);
    const int num_blocks = num_blocks_m * num_blocks_n;
    constexpr int group_size_m = 16/CLUSTER_M;
    constexpr int group_size_n = 8/CLUSTER_N;
    constexpr int tiles_in_group = group_size_m * group_size_n;

    int wg_idx = threadIdx.x / 128;
    const int tid = threadIdx.x % 128;

    if (threadIdx.x == 0) {
        for (int i = 0; i < PIPE; ++i) {
            init_barriers(&full_barrier[i], 1);
            init_barriers(&empty_barrier[i], num_consumers * CLUSTERS);
        }
    }
    asm volatile("barrier.cluster.arrive;\n" : :);
    asm volatile("barrier.cluster.wait;\n" : :);

    asm volatile("mov.u32 %0, %cluster_ctarank;\n" : "=r"(rank) :);
    uint32_t rank_m = rank / CLUSTER_N;
    uint32_t rank_n = rank % CLUSTER_N;

    if (wg_idx == 0) {
        constexpr int num_regs = (num_consumers <= 2 ? 24 : 32);
        warpgroup_reg_dealloc<num_regs>();
        
        if (tid == 0) {
            int pipe_lane = 0;
            int p = 0;
            uint32_t col_mask = 0;
            for (int i = 0; i < CLUSTER_M; ++i) {
                col_mask |= (1 << (i * CLUSTER_N));
            }
            int tile_m, tile_n;

            for (int tile_idx = cluster_id; tile_idx < num_blocks; tile_idx+=NUM_SM/CLUSTERS) {
                calculate_tile_indices(tile_idx, num_blocks_n, group_size_m, group_size_n, tiles_in_group, tile_m, tile_n);
                tile_m = tile_m * CLUSTER_M + rank_m;
                tile_n = tile_n * CLUSTER_N + rank_n;

                for (int k_tile = 0; k_tile < num_tiles_k; ++k_tile, ++pipe_lane) {
                    if (pipe_lane == PIPE) {pipe_lane = 0; p ^= 1; }
                    wait(&empty_barrier[pipe_lane], p);

                    expect_bytes(&full_barrier[pipe_lane], (BK*BN+BK*BM)*sizeof(bf16));
                    if constexpr (CLUSTER_N > 1) {
                        uint32_t mask = ((1 << CLUSTER_N) - 1) << (rank_m * CLUSTER_N);
                        if (rank_n == 0) {
                            load_async_multi(&sA[pipe_lane*BM*BK], &tensorMapA, &full_barrier[pipe_lane], k_tile*BK, tile_m*BM, mask);
                        }
                    } else {
                        load_async_3d(&sA[pipe_lane*BM*BK], &tensorMapA, &full_barrier[pipe_lane], k_tile*BK, tile_m*BM);
                    }

                    if constexpr (CLUSTER_M > 1) {
                        if (rank_m == 0) {
                            load_async_multi(&sB[pipe_lane*BM*BK], &tensorMapB, &full_barrier[pipe_lane], k_tile*BK, tile_n*BN, col_mask);
                        }
                    } else {
                        load_async_3d(&sB[pipe_lane*BN*BK], &tensorMapB, &full_barrier[pipe_lane], k_tile*BK, tile_n*BN);
                    }
                    
                }
            }
        }
      }
      else {
        constexpr int num_regs = (num_consumers == 1 ? 256 : (num_consumers == 2 ? 240 : 160));
        warpgroup_reg_alloc<num_regs>();
        float d[B_WG_M/WGMMA_M][WGMMA_N/16][8];
        
        --wg_idx;

        for (int i = 0; i < PIPE; ++i) {
            if (tid == 0) arrive(&empty_barrier[i], 1);
        }
        int pipe_lane = 0;
        int p = 0;
        int tile_m, tile_n;
        for (int tile_idx = blockIdx.x; tile_idx < num_blocks; tile_idx+=NUM_SM) {
            calculate_tile_indices(tile_idx, num_blocks_n, group_size_m, group_size_n, tiles_in_group, tile_m, tile_n);
            tile_m = tile_m * CLUSTER_M + rank_m;
            tile_n = tile_n * CLUSTER_N + rank_n;
            memset(d, 0, sizeof(d));
            for (int k_tile = 0; k_tile < num_tiles_k; ++k_tile, ++pipe_lane) {
                if (pipe_lane == PIPE) {pipe_lane = 0; p ^= 1; }
                wait(&full_barrier[pipe_lane], p);
                warpgroup_arrive();
                #pragma unroll
                for (int m_it = 0; m_it < B_WG_M / WGMMA_M; ++m_it) {
                    bf16 *wgmma_sA = sA + pipe_lane*BM*BK + BK*WGMMA_M*(m_it + wg_idx*(B_WG_M/WGMMA_M));
                    bf16 *wgmma_sB = sB + pipe_lane*BK*BN;
                    #pragma unroll
                    for (int k_it = 0; k_it < BK / WGMMA_K; ++k_it) {
                        wgmma<WGMMA_N, 1, 1, 1, 0, 0>(d[m_it], &wgmma_sA[k_it*WGMMA_K], &wgmma_sB[k_it*WGMMA_K]);
                    }
                }
                warpgroup_commit_batch();
                warpgroup_wait<0>();
                if (tid < CLUSTERS) arrive_cluster(&empty_barrier[pipe_lane], tid);
            }
        

            int lane = tid % 32;
            int warp = tid / 32;
            int row = warp*16 + lane / 4;
            bf16 *block_C = C + tile_n*BN*M + tile_m*BM;

            #pragma unroll
            for (int m_it = 0; m_it < B_WG_M / WGMMA_M; ++m_it) {
                int yo = m_it*WGMMA_M + wg_idx*B_WG_M;
                #pragma unroll
                for (int w = 0; w < WGMMA_N/16; ++w) {
                    int col = 16*w + 2*(tid % 4);
                    #define IDX(i, j) ((j)*M + ((i) + yo))
                    block_C[IDX(row,         col    )] = d[m_it][w][0];
                    block_C[IDX(row,         col + 1)] = d[m_it][w][1];
                    block_C[IDX(row + 8,     col    )] = d[m_it][w][2];
                    block_C[IDX(row + 8,     col + 1)] = d[m_it][w][3];
                    block_C[IDX(row,         col + 8)] = d[m_it][w][4];
                    block_C[IDX(row,         col + 9)] = d[m_it][w][5];
                    block_C[IDX(row + 8,     col + 8)] = d[m_it][w][6];
                    block_C[IDX(row + 8,     col + 9)] = d[m_it][w][7];
                    #undef IDX
                }
            }
        }
    }
}

void matmul_v7(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    constexpr int BM = 64*2;
    constexpr int BN = 256;
    constexpr int BK = 64;
    constexpr int NUM_THREADS = 128*3;
    constexpr int PIPE = 3;
    constexpr int CLUSTER_M = 2;
    constexpr int CLUSTER_N = 1;

    constexpr int NUM_SM = 128;
    d_tma_map_A = create_tensor_map<BM, BK>(A, M, K);
    d_tma_map_B = create_tensor_map<BN, BK>(B, N, K);

    auto* kernel = matmul_kernel_v7<BM,BN,BK,NUM_THREADS,PIPE,NUM_SM,CLUSTER_M,CLUSTER_N>;
    size_t smem_size = sizeof(SharedStorage<BM, BN, BK, PIPE>);
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    kernel<<<NUM_SM, NUM_THREADS, smem_size>>>(M, N, K, C, d_tma_map_A, d_tma_map_B);
}

}

using M7::matmul_v7;
