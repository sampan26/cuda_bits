#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctime>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_bf16.h>
#include <cassert>
#include <unistd.h>

typedef __hip_bfloat16 bf16;
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line, hipGetErrorString(error));
    exit(1);
  }
}
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

// Simple matrix multiplication kernel as placeholder
__global__ void matmul_v1_kernel(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += __bfloat162float(A[row * K + k]) * __bfloat162float(B[k * N + col]);
        }
        C[row * N + col] = __float2bfloat16(sum);
    }
}

void matmul_v1(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    dim3 block(16, 16);
    dim3 grid(CEIL_DIV(N, block.x), CEIL_DIV(M, block.y));
    matmul_v1_kernel<<<grid, block>>>(M, N, K, A, B, C);
}

std::default_random_engine generator(69);

hipblasHandle_t cublas_handle;
void runCublasGemmBF16(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    const float alpha = 1.0f, beta = 0.0f;
    hipblasStatus_t status = hipblasGemmEx(cublas_handle, 
                                        HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                        M, N, K, 
                                        &alpha, 
                                        A, HIP_R_16BF, M,
                                        B, HIP_R_16BF, K, 
                                        &beta, 
                                        C, HIP_R_16BF, M,
                                        HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS error: " << status << std::endl;
        exit(1);
    }
}

void run_kernel(int kernel_num, int M, int N, int K, bf16 *A, bf16 *B, bf16 *C, int *DB = nullptr) {
    switch (kernel_num) {
        case 0:
            runCublasGemmBF16(M, N, K, A, B, C);
            break;
        case 1:
            matmul_v1(M, N, K, A, B, C);
            break;
    }
}
    
int yo = 0;
void randomize_matrix(bf16 *mat, int N) {
    std::normal_distribution<float> distribution(0, 1);
    for (int i = 0; i < N; i++) {
        mat[i] = __float2bfloat16(distribution(generator));
    }
    ++yo;
}

bool verify_matrix(bf16 *matRef, bf16 *matOut, int N) {
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++) {
        int r = i / 8192, c = i % 8192;
        int it = c*8192+r;
        diff = std::fabs(__bfloat162float(matRef[i]) - __bfloat162float(matOut[i]));
        if (diff > 0.1) {
            printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
                   __bfloat162float(matRef[i]), __bfloat162float(matOut[i]), diff, i);
            return false;
        }
    }
    return true;
}

int main() {
    hipblasCreate(&cublas_handle);

    long max_size = 8192;
    long m = max_size, n = max_size, k = max_size;

    bf16 *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;  // host matrices
    bf16 *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr; // device matrices
    
    int *DB = nullptr; 
    int *dDB = nullptr;  

    A = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    B = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    C = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    C_ref = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    DB = (int *)malloc(sizeof(int) * max_size * 128);
    
    cudaCheck(hipMalloc((void **)&dDB, sizeof(int) * max_size * 128));
    
    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);

    cudaCheck(hipMalloc((void **)&dA, sizeof(bf16) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dB, sizeof(bf16) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dC, sizeof(bf16) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(bf16) * max_size * max_size));
    
    cudaCheck(hipMemcpy(dA, A, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));

    for (int kernel_num : {0, 1}) {
        // Give the GPU some rest to avoid thermal throttling
        sleep(5);
        std::cout << "KERNEL " << kernel_num << std::endl;

        memset(C, 0, sizeof(bf16) * max_size * max_size);
        cudaCheck(hipMemcpy(dC, C, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(dC_ref, C, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
        memset(DB, ~0, sizeof(int) * max_size * 128);
        cudaCheck(hipMemcpy(dDB, DB, sizeof(int) * max_size * 128, hipMemcpyHostToDevice));
        
        run_kernel(0, m, n, k, dA, dB, dC_ref); // cuBLAS
        run_kernel(kernel_num, m, n, k, dA, dB, dC, dDB); // Executes the kernel, modifies the result matrix
        
        cudaCheck(hipDeviceSynchronize());
        cudaCheck(hipGetLastError()); // Check for async errors during kernel run
        
        hipMemcpy(C, dC, sizeof(bf16) * max_size * max_size, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(bf16) * max_size * max_size, hipMemcpyDeviceToHost);

        if (kernel_num > 0 && !verify_matrix(C_ref, C, m * n)) {
            std::cout << "~~~~~~~~~~~~~~~~ Failed to pass the correctness verification against cuBLAS. ~~~~~~~~~~~~~~~~" << std::endl;
            hipMemcpy(DB, dDB, sizeof(int) * max_size * 8, hipMemcpyDeviceToHost);
            printf("%f\n", __bfloat162float(C_ref[m]));
        } else if (kernel_num > 0) {
            std::cout << "Correctness verification passed!" << std::endl;
        }
    }
    
    // Cleanup
    free(A);
    free(B);
    free(C);
    free(C_ref);
    free(DB);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);
    hipFree(dDB);
    hipblasDestroy(cublas_handle);
    
    return 0;
}
