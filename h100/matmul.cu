#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctime>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_bf16.h>
#include <cassert>
#include <unistd.h>

#include "src/matmul_v1.cu"
#include "src/matmul_v2.cu"


typedef __hip_bfloat16 bf16;
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line, hipGetErrorString(error));
    exit(1);
  }
}
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

std::default_random_engine generator(69);

hipblasHandle_t cublas_handle;
void runCublasGemmBF16(int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    float alpha = 1, beta = 0;
    // C(column major) = A(row major) * B(column major)
    hipblasStatus_t status = hipblasGemmEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, A, HIP_R_16BF,
      N, B, HIP_R_16BF, K, &beta, C, HIP_R_16BF, N, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
  
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cout << "CUBLAS error: " << status << std::endl;
      exit(1);
    }
}

void run_kernel(int kernel_num, int M, int N, int K, bf16 *A, bf16 *B, bf16 *C) {
    switch (kernel_num) {
        case 0:
            runCublasGemmBF16(M, N, K, A, B, C);
            break;
        case 1:
            matmul_v1(M, N, K, A, B, C);
            break;
        case 2:
            matmul_v2(M, N, K, A, B, C);
            break;
    }
}
    
int yo = 0;
void randomize_matrix(bf16 *mat, int N) {
  std::normal_distribution<float> distribution(0, 1);
  for (int i = 0; i < N; i++) {
    mat[i] = distribution(generator);
  }
  ++yo;
}

bool verify_matrix(bf16 *matRef, bf16 *matOut, int N) {
    double diff = 0.0;
    int i;
    for (i = 0; i < N; i++) {
        int r = i / 8192, c = i % 8192;
        int it = c*8192+r;
        diff = std::fabs(__bfloat162float(matRef[i]) - __bfloat162float(matOut[i]));
        if (diff > 0.1) {
            printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
                   __bfloat162float(matRef[i]), __bfloat162float(matOut[i]), diff, i);
            return false;
        }
    }
    return true;
}

int main() {
    hipblasCreate(&cublas_handle);
    float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    long max_size = 8192;
    long m = max_size, n = max_size, k = max_size;

    bf16 *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;  // host matrices
    bf16 *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr; // device matrices
    
    A = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    B = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    C = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    C_ref = (bf16 *)malloc(sizeof(bf16) * max_size * max_size);
    
    
    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);

    cudaCheck(hipMalloc((void **)&dA, sizeof(bf16) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dB, sizeof(bf16) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dC, sizeof(bf16) * max_size * max_size));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(bf16) * max_size * max_size));
    
    cudaCheck(hipMemcpy(dA, A, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));

    for (int kernel_num : {0, 1}) {
        // Give the GPU some rest to avoid thermal throttling
        sleep(5);
        std::cout << "KERNEL " << kernel_num << std::endl;

        memset(C, 0, sizeof(bf16) * max_size * max_size);
        cudaCheck(hipMemcpy(dC, C, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(dC_ref, C, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
        
        run_kernel(0, m, n, k, dA, dB, dC_ref); // cuBLAS
        run_kernel(kernel_num, m, n, k, dA, dB, dC); // Executes the kernel, modifies the result matrix
        
        cudaCheck(hipDeviceSynchronize());
        cudaCheck(hipGetLastError()); // Check for async errors during kernel run
        
        hipMemcpy(C, dC, sizeof(bf16) * max_size * max_size, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(bf16) * max_size * max_size, hipMemcpyDeviceToHost);

        if (kernel_num > 0 && !verify_matrix(C_ref, C, m * n)) {
            std::cout << "~~~~~~~~~~~~~~~~ Failed to pass the correctness verification against cuBLAS. ~~~~~~~~~~~~~~~~" << std::endl;
            printf("%f\n", __bfloat162float(C_ref[m]));
        } else if (kernel_num > 0) {
            std::cout << "Correctness verification passed!" << std::endl;
        }

        hipEventRecord(start);
        for (int j = 0; j < 8; ++j) {
          run_kernel(kernel_num, m, n, k, dA, dB, dC);
        }
        hipEventRecord(stop);
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        
        long flops = (2LL * m) * (n * k);
        printf(
            "Average elapsed time: (%7.6f) s, performance: (%7.1f) TFLOPS. size: (%ld).\n\n",
            elapsed_time / 1000.0 / 8,
            (8 * flops * 1e-9) / elapsed_time, m);
        
    }
    
    // Cleanup
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);
    hipblasDestroy(cublas_handle);
    
    return 0;
}
