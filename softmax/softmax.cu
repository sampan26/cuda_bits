
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

__host__ __device__ constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }

__global__ void softmax_kernel_v1(const float* input, float* output, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= M) return;

    float max_val = -INFINITY;
    for (int col = 0; col < N; col++) {
        max_val = fmaxf(max_val, input[row * N + col]);
    }

    float denom = 0.0f;
    for (int col = 0; col < N; col++) {
        denom += expf(input[row * N + col] - max_val);
    }

    for (int col = 0; col < N; col++) {
        output[row * N + col] = expf(input[row * N + col] - max_val) / denom;
    }
}

void softmax_v1(const float* input, float* output, int M, int N) {
    const int BLOCK_SIZE = 1024;
    int grid_size = cdiv(M, BLOCK_SIZE);
    softmax_kernel_v1<<<grid_size, BLOCK_SIZE>>>(input, output, M, N);
}

__global__ void softmax_kernel_v2(const float* input, float* output, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= M) return;

    float max_val = -INFINITY;
    float norm = 0.0f;

    for (int col = 0; col < N; col++) {
        float cur_val = input[row * N + col];
        if (cur_val > max_val) {
            norm = norm * expf(max_val - cur_val);
            max_val = cur_val;
        }
        norm += expf(cur_val - max_val);
    }

    for (int col = 0; col < N; col++) {
        output[row * N + col] = expf(input[row * N + col] - max_val) / norm;
    }
}

void softmax_v2(const float* input, float* output, int M, int N) {
    const int BLOCK_SIZE = 1024;
    int grid_size = cdiv(M, BLOCK_SIZE);
    softmax_kernel_v2<<<grid_size, BLOCK_SIZE>>>(input, output, M, N);
}