
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <assert.h>

__host__ __device__ constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }

constexpr int WARP_SIZE = 32;


// naive kernel. 1 row dot 1 column
// to compute 1 output element:
// - load 1 row from A (1xK) and 1 column from B (Kx1)
// - K multiplications and (K-1) additions
// => arithmetic intensity ~ O(1)
__global__ void matmul_v1_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= M || col >= N) return;

    // broadcast read from A since each warp reads the same A value
    // coalesce read from B since each warp reads consecutive B values
    float total = 0.0f;
    for (int k = 0; k < K; k++) {
        total += A[row * K + k] * B[k * N + col]; 
    }
    // coalesce write to C since each warp writes consecutive C values
    C[row * N + col] = total;
}

void matmul_v1(const float *A, const float *B, float *C, int M, int N, int K) {
    int block_size_total;
    int min_grid_size;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size_total, matmul_v1_kernel, 0, 0);

    // NOTE: blockDim.x must be a multiple of 32 (warpSize) to ensure coalesce memory access
    int BLOCK_X = WARP_SIZE;
    int BLOCK_Y = block_size_total / WARP_SIZE;
    dim3 block_size(BLOCK_X, BLOCK_Y);
    dim3 grid_size(cdiv(N, BLOCK_X), cdiv(M, BLOCK_Y));
    matmul_v1_kernel<<<grid_size, block_size>>>(A, B, C, M, N, K);
}

// thread-block tiling: read 2D block into shared memory for caching
// to compute BLOCK_SIZExBLOCK_SIZE output elements:
// - load BLOCK_SIZExBLOCK_SIZE of A and B from global memory
// - amount of compute is unchanged
// => arithmetic intensity ~ O(BLOCK_SIZE)
template <int BLOCK_SIZE>
__global__ void matmul_v2_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int tid_x = threadIdx.x;
    const int tid_y = threadIdx.y;

    const int offset_m = blockIdx.y * BLOCK_SIZE;
    const int offset_n = blockIdx.x * BLOCK_SIZE;

    A += offset_m * K;
    B += offset_n;
    C += offset_m * K + offset_n;

    __shared__ float A_shmem[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_shmem[BLOCK_SIZE][BLOCK_SIZE];
    float acc = 0.0f;
    
    for (int offset_k = 0; offset_k < K; offset_k+=BLOCK_SIZE) {
        // load data from global memory (DDR/HBM) to shared memory (SRAM)
        // notice now each thread only loads 2 x n_blocks elements
        // coalesced memory read for both A and B
       A_shmem[tid_y][tid_x] = tid_y < (M - offset_m) && tid_x < (K - offset_k) ? A[tid_y * K + tid_x] : 0.0f;
       B_shmem[tid_y][tid_x] = tid_y < (K - offset_k) && tid_x < (N - offset_n) ? B[tid_y * N + tid_x] : 0.0f;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++) {
            acc += A_shmem[tid_y][k] * B_shmem[k][tid_x];
        }

        __syncthreads();

        A += BLOCK_SIZE;
        B += BLOCK_SIZE * N;
    }

    if (tid_y < (M - offset_m) && tid_x < (N - offset_n))
        C[tid_y * N + tid_x] = acc;
}

void matmul_v2(const float *A, const float *B, float *C, int M, int N, int K) {
    constexpr int BLOCK_SIZE = 32;
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size(cdiv(N, BLOCK_SIZE), cdiv(M, BLOCK_SIZE));
    matmul_v2_kernel<BLOCK_SIZE><<<grid_size, block_size>>>(A, B, C, M, N, K);
}

template <int BLOCK_SIZE, int HEIGHT, int WIDTH>
__device__ void load_shmem(const float *in, int in_row_stride, int in_max_row, int in_max_col,
                           float out[HEIGHT][WIDTH], int tid) {
    for (int idx = tid; idx < HEIGHT * WIDTH; idx += BLOCK_SIZE) {
        const int row = idx / WIDTH;
        const int col = idx % WIDTH;
        out[row][col] = row < in_max_row && col < in_max_col ? in[row * in_row_stride + col] : 0.0f;
    }
}

template <int BLOCK_SIZE, int BLOCK_M, int BLOCK_N, int BLOCK_K>
__global__ void matmul_v3_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int tid = threadIdx.x;
    const int block_id = blockIdx.x;

    const int num_blocks_per_row = cdiv(N, BLOCK_N);
    const int block_id_m = block_id / num_blocks_per_row;
    const int block_id_n = block_id % num_blocks_per_row;

    const int offset_m = block_id_m * BLOCK_M;
    const int offset_n = block_id_n * BLOCK_N;

    A += offset_m * K;
    B += offset_n;
    C += offset_m * N + offset_n;

    __shared__ float A_shmem[BLOCK_M][BLOCK_K];
    __shared__ float B_shmem[BLOCK_K][BLOCK_N];

    static_assert((BLOCK_M * BLOCK_N) % BLOCK_SIZE == 0);
    float acc[BLOCK_M * BLOCK_N / BLOCK_SIZE] = {0.0f};

    for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
        load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
        load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);

        __syncthreads();

        for (int idx = tid; idx < BLOCK_M * BLOCK_N; idx+=BLOCK_SIZE) {
            const int local_idx = idx / BLOCK_SIZE;
            const int row = idx / BLOCK_N;
            const int col = idx % BLOCK_N;

            for (int k = 0; k < BLOCK_K; k++) {
                acc[local_idx] += A_shmem[row][k] * B_shmem[k][col];
            }
        }
        __syncthreads();

        A += BLOCK_K;
        B += BLOCK_K * N;
    }
    
    for (int idx = tid; idx < BLOCK_M * BLOCK_N; idx+=BLOCK_SIZE) {
        const int local_idx = idx / BLOCK_SIZE;
        const int row = idx / BLOCK_N;
        const int col = idx % BLOCK_N;

        if (row < (M - offset_m) && col < (N - offset_n)) {
            C[row * N + col] = acc[local_idx];
        }
    }
}

void matmul_v3(const float *A, const float *B, float *C, int M, int N, int K) {
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
    const int BLOCK_SIZE = 256;
    const int grid_size = cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N);
    matmul_v3_kernel<BLOCK_SIZE, BLOCK_M, BLOCK_N, BLOCK_K><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

