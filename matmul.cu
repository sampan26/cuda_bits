
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

__host__ __device__ constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }
constexpr bool is_power_of_two(int x) { return x > 0 && (x & (x - 1)) == 0; }  // https://stackoverflow.com/a/1804686
constexpr int WARP_SIZE = 32;


// naive kernel. 1 row dot 1 column
// to compute 1 output element:
// - load 1 row from A (1xK) and 1 column from B (Kx1)
// - K multiplications and (K-1) additions
// => arithmetic intensity ~ O(1)
__global__ void matmul_v1_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= M || col >= N) return;

    // broadcast read from A since each warp reads the same A value
    // coalesce read from B since each warp reads consecutive B values
    float total = 0.0f;
    for (int k = 0; k < K; k++) {
        total += A[row * K + k] * B[k * N + col]; 
    }
    // coalesce write to C since each warp writes consecutive C values
    C[row * N + col] = total;
}

void matmul_v1(const float *A, const float *B, float *C, int M, int N, int K) {
    int block_size_total;
    int min_grid_size;
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size_total, matmul_v1_kernel, 0, 0);

    // NOTE: blockDim.x must be a multiple of 32 (warpSize) to ensure coalesce memory access
    int BLOCK_X = WARP_SIZE;
    int BLOCK_Y = block_size_total / WARP_SIZE;
    dim3 block_size(BLOCK_X, BLOCK_Y);
    dim3 grid_size(cdiv(N, BLOCK_X), cdiv(M, BLOCK_Y));
    matmul_v1_kernel<<<grid_size, block_size>>>(A, B, C, M, N, K);
}

// thread-block tiling: read 2D block into shared memory for caching
// to compute BLOCK_SIZExBLOCK_SIZE output elements:
// - load BLOCK_SIZExBLOCK_SIZE of A and B from global memory
// - amount of compute is unchanged
// => arithmetic intensity ~ O(BLOCK_SIZE)
template <int BLOCK_SIZE>
__global__ void matmul_v2_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int tid_x = threadIdx.x;
    const int tid_y = threadIdx.y;

    const int offset_m = blockIdx.y * BLOCK_SIZE;
    const int offset_n = blockIdx.x * BLOCK_SIZE;

    A += offset_m * K;
    B += offset_n;
    C += offset_m * K + offset_n;

    __shared__ float A_shmem[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_shmem[BLOCK_SIZE][BLOCK_SIZE];
    float acc = 0.0f;
    
    for (int offset_k = 0; offset_k < K; offset_k+=BLOCK_SIZE) {
        // load data from global memory (DDR/HBM) to shared memory (SRAM)
        // notice now each thread only loads 2 x n_blocks elements
        // coalesced memory read for both A and B
       A_shmem[tid_y][tid_x] = tid_y < (M - offset_m) && tid_x < (K - offset_k) ? A[tid_y * K + tid_x] : 0.0f;
       B_shmem[tid_y][tid_x] = tid_y < (K - offset_k) && tid_x < (N - offset_n) ? B[tid_y * N + tid_x] : 0.0f;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++) {
            acc += A_shmem[tid_y][k] * B_shmem[k][tid_x];
        }

        __syncthreads();

        A += BLOCK_SIZE;
        B += BLOCK_SIZE * N;
    }

    if (tid_y < (M - offset_m) && tid_x < (N - offset_n))
        C[tid_y * N + tid_x] = acc;
}

void matmul_v2(const float *A, const float *B, float *C, int M, int N, int K) {
    constexpr int BLOCK_SIZE = 32;
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size(cdiv(N, BLOCK_SIZE), cdiv(M, BLOCK_SIZE));
    matmul_v2_kernel<BLOCK_SIZE><<<grid_size, block_size>>>(A, B, C, M, N, K);
}

// we want to load a (HEIGHT, WIDTH) tile from global to shared memory.
// just load a BLOCK_SIZE of data until the whole tile is loaded.
template <int BLOCK_SIZE, int HEIGHT, int WIDTH>
__device__ void load_shmem(const float *in, int in_row_stride, int in_max_row, int in_max_col,
                           float out[HEIGHT][WIDTH], int tid) {
    for (int idx = tid; idx < HEIGHT * WIDTH; idx += BLOCK_SIZE) {
        const int row = idx / WIDTH;
        const int col = idx % WIDTH;
        out[row][col] = row < in_max_row && col < in_max_col ? in[row * in_row_stride + col] : 0.0f;
    }
}

// thread coarsening
template <int BLOCK_SIZE, int BLOCK_M, int BLOCK_N, int BLOCK_K>
__global__ void matmul_v3_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    const int tid = threadIdx.x;
    const int block_id = blockIdx.x;

    // assign block linearly
    const int num_blocks_per_row = cdiv(N, BLOCK_N);
    const int block_id_m = block_id / num_blocks_per_row;
    const int block_id_n = block_id % num_blocks_per_row;

    const int offset_m = block_id_m * BLOCK_M;
    const int offset_n = block_id_n * BLOCK_N;

    A += offset_m * K;
    B += offset_n;
    C += offset_m * N + offset_n;

    __shared__ float A_shmem[BLOCK_M][BLOCK_K];
    __shared__ float B_shmem[BLOCK_K][BLOCK_N];

    // each thread is responsible for (BLOCK_M * BLOCK_N / BLOCK_SIZE) output elements
    static_assert((BLOCK_M * BLOCK_N) % BLOCK_SIZE == 0);
    float acc[BLOCK_M * BLOCK_N / BLOCK_SIZE] = {0.0f};

    // we move block by block along K dim
    for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
        // decouple global memory read, so we don't need to care about assigning which thread to read which element.
        // load (BLOCK_M, BLOCK_K) from A and (BLOCK_K, BLOCK_N) from B
        load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
        load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);

        __syncthreads();

         // do a mini matmul of (BLOCK_M, BLOCK_K) x (BLOCK_K, BLOCK_N) = (BLOCK_M, BLOCK_N)
        // simply assign a BLOCK_SIZE of threads to a BLOCK_SIZE of elements in output tile
        // there is shared memory bank conflict
        for (int idx = tid; idx < BLOCK_M * BLOCK_N; idx+=BLOCK_SIZE) {
            const int local_idx = idx / BLOCK_SIZE;
            const int row = idx / BLOCK_N;
            const int col = idx % BLOCK_N;

            for (int k = 0; k < BLOCK_K; k++) {
                acc[local_idx] += A_shmem[row][k] * B_shmem[k][col];
            }
        }
        __syncthreads();

        A += BLOCK_K;
        B += BLOCK_K * N;
    }
    
    // write (BLOCK_M, BLOCK_N) to C
    for (int idx = tid; idx < BLOCK_M * BLOCK_N; idx+=BLOCK_SIZE) {
        const int local_idx = idx / BLOCK_SIZE;
        const int row = idx / BLOCK_N;
        const int col = idx % BLOCK_N;

        if (row < (M - offset_m) && col < (N - offset_n)) {
            C[row * N + col] = acc[local_idx];
        }
    }
}

void matmul_v3(const float *A, const float *B, float *C, int M, int N, int K) {
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
    const int BLOCK_SIZE = 256;
    const int grid_size = cdiv(M, BLOCK_M) * cdiv(N, BLOCK_N);
    matmul_v3_kernel<BLOCK_SIZE, BLOCK_M, BLOCK_N, BLOCK_K><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

// 2D thread-tiling with register cache
// each thread will calculate (THREAD_M, THREAD_N) thread-tile of output (BLOCK_M, BLOCK_N) block-tile
template <int BLOCK_M, int BLOCK_N, int BLOCK_K, int THREAD_M, int THREAD_N>
__global__ void matmul_v4_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    static_assert(BLOCK_M % THREAD_M == 0);
    static_assert(BLOCK_N % THREAD_N == 0);
    constexpr int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (THREAD_M * THREAD_N);
    const int tid = threadIdx.x;
    const int block_id = blockIdx.x;

    const int num_blocks_per_row = cdiv(N, BLOCK_N);
    const int block_id_m = block_id / num_blocks_per_row;
    const int block_id_n = block_id % num_blocks_per_row;
    const int offset_m = block_id_m * BLOCK_M;
    const int offset_n = block_id_n * BLOCK_N;

    const int num_threads_per_row = BLOCK_N / THREAD_N;
    const int tile_thread_id_m = tid / num_threads_per_row;
    const int tile_thread_id_n = tid % num_threads_per_row;
    const int tile_thread_offset_m = tile_thread_id_m * THREAD_M;
    const int tile_thread_offset_n = tile_thread_id_n * THREAD_N;

    __shared__ float A_shmem[BLOCK_M][BLOCK_K];
    __shared__ float B_shmem[BLOCK_K][BLOCK_N];
    float acc[THREAD_M][THREAD_N] = {0.0f};

    A += offset_m * K;
    B += offset_n;
    
    const float *A_thread_tile = reinterpret_cast<const float *>(A_shmem) + tile_thread_offset_m * BLOCK_K;
    const float *B_thread_tile = reinterpret_cast<const float *>(B_shmem) + tile_thread_offset_n;

    for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
        load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
        load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);
        __syncthreads();

        // mini-matmul with thread-tile. same structure as block-tile.
        for (int k = 0; k < BLOCK_K; k++) {
            float A_reg[THREAD_M];
            float B_reg[THREAD_N]; // register cache

            // load data from shared memory to registers
            // there is shared memory bank conflict
            for (int m = 0; m < THREAD_M; m++)
                A_reg[m] = A_thread_tile[m * BLOCK_K + k];

            for (int n = 0; n < THREAD_N; n++)
                B_reg[n] = B_thread_tile[k * BLOCK_N + n];

            // for each (THREAD_M, THEAD_N) output, we only need to read
                // (THREAD_M, BLOCK_K) of A and (BLOCK_K, THREAD_N) for B from shared memory.
            for (int m = 0; m < THREAD_M; m++ ) {
                for (int n = 0; n < THREAD_N; n++) {
                    acc[m][n] += A_reg[m] * B_reg[n];
                }
            }
        }
        __syncthreads();

        A += BLOCK_K;
        B += BLOCK_K * N;
    }

    C += (offset_m + tile_thread_offset_m) * N + (offset_n + tile_thread_offset_n);

    // uncoalesced memory write
    // fixing it doesn't seem to make the kernel faster.
    // vectorized write is slower.
    for (int m = 0; m < THREAD_M; m++) {
        for (int n = 0; n < THREAD_N; n++) {
            if (m < (M - (offset_m + tile_thread_offset_m)) && n < (N - (offset_n + tile_thread_offset_n)))
                C[m * N + n] = acc[m][n];
        }
    }
}

void matmul_v4(const float *A, const float *B, float *C, int M, int N, int K) {
    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
    const int THREAD_M = 8, THREAD_N = 8;
    const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (THREAD_M * THREAD_N);  // 256
    const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
    matmul_v4_kernel<BLOCK_M, BLOCK_N, BLOCK_K, THREAD_M, THREAD_N><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

// 2D warp-tiling with register cache
// we partition output tile (BLOCK_M, BLOCK_N) into tiles of (WARP_M, WARP_N)
// we use the exact number of warps in a threadblock as the number of warp tiles
// for each output warp tile (WARP_M, WARP_N), we further divide it into MMA tiles (MMA_M, MMA_N)
// for each MMA tile (MMA_M, MMA_N), we divide it exactly to 32 thread tiles (THREAD_M, THREAD_N),
// since there are 32 threads in a warp.
template <int BLOCK_M, int BLOCK_N, int BLOCK_K, int WARP_M, int WARP_N, int MMA_M, int MMA_N, int THREAD_N>
__global__ void matmul_v5_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    static_assert(BLOCK_M % WARP_M == 0);
    static_assert(BLOCK_N % WARP_N == 0);
    static_assert(WARP_M % MMA_M == 0);
    static_assert(WARP_N % MMA_N == 0);
    static_assert((MMA_M * MMA_N / THREAD_N) % WARP_SIZE == 0);

    constexpr int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_SIZE * WARP_N) * WARP_SIZE;
    constexpr int NUM_MMA_M = WARP_M / MMA_M;
    constexpr int NUM_MMA_N = WARP_N / MMA_N;
    constexpr int THREAD_M = MMA_M * MMA_N / (WARP_SIZE * THREAD_N);

    const int tid = threadIdx.x;
    const int block_id = blockIdx.x;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;

    const int num_blocks_per_row = cdiv(N, BLOCK_N);
    const int block_id_m = block_id / num_blocks_per_row;
    const int block_id_n = block_id % num_blocks_per_row;
    const int offset_m = block_id_m * BLOCK_M;
    const int offset_n = block_id_n * BLOCK_N;

    constexpr int num_warps_per_row = BLOCK_N / WARP_N;
    const int warp_id_m = warp_id / num_warps_per_row;
    const int warp_id_n = warp_id % num_warps_per_row;
    const int warp_tile_offset_m = warp_id_m * WARP_M;
    const int warp_tile_offset_n = warp_id_n * WARP_N;

    constexpr int num_threads_per_row = MMA_N / THREAD_N;
    const int tile_thread_id_m = lane_id / num_threads_per_row;
    const int tile_thread_id_n = lane_id % num_threads_per_row;
    const int tile_thread_offset_m = tile_thread_id_m * THREAD_M;
    const int tile_thread_offset_n = tile_thread_id_n * THREAD_N;

    __shared__ float A_shmem[BLOCK_M][BLOCK_K];
    __shared__ float B_shmem[BLOCK_K][BLOCK_N];
    float acc[NUM_MMA_M][NUM_MMA_N][THREAD_M][THREAD_N] = {0.0f};

    A += offset_m * K;
    B += offset_n;
    

    const float *A_thread_tile = reinterpret_cast<const float *>(A_shmem) + (tile_thread_offset_m + warp_tile_offset_m) * BLOCK_K;
    const float *B_thread_tile = reinterpret_cast<const float *>(B_shmem) + (tile_thread_offset_n + warp_tile_offset_n);

    // points to the corresponding thread tile of the current thread in the first MMA tile
    for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
        load_shmem<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, M - offset_m, K - offset_k, A_shmem, tid);
        load_shmem<BLOCK_SIZE, BLOCK_K, BLOCK_N>(B, N, K - offset_k, N - offset_n, B_shmem, tid);
        __syncthreads();
        
        // implicit WARP_K = MMA_K = THREAD_K = 1
        for (int k = 0; k < BLOCK_K; k++) {
            float A_reg[NUM_MMA_M][THREAD_M]; // 2Dregister cache
            float B_reg[NUM_MMA_N][THREAD_N]; 

            // notice we have extra loops to iterate over MMA tiles
            for (int mm_tile_id_m = 0; mm_tile_id_m < NUM_MMA_M; mm_tile_id_m++)
                for (int m = 0; m < THREAD_M; m++)
                    A_reg[mm_tile_id_m][m] = A_thread_tile[(mm_tile_id_m * MMA_M + m) * BLOCK_K + k];

            for (int mm_tile_id_n = 0; mm_tile_id_n < NUM_MMA_N; mm_tile_id_n++)
                for (int n = 0; n < THREAD_N; n++)
                    B_reg[mm_tile_id_n][n] = B_thread_tile[k * BLOCK_N + (mm_tile_id_n * MMA_N + n)];

           
             for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
                for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++)
                    for (int m = 0; m < THREAD_M; m++)
                        for (int n = 0; n < THREAD_N; n++)
                            acc[mma_tile_id_m][mma_tile_id_n][m][n] += A_reg[mma_tile_id_m][m] * B_reg[mma_tile_id_n][n];
        }
        __syncthreads();

        A += BLOCK_K;
        B += BLOCK_K * N;
    }

    // points to the corresponding thread tile in the first MMA tile
    int C_offset_m = offset_m + tile_thread_offset_m + warp_tile_offset_m;
    int C_offset_n = offset_n + tile_thread_offset_n + warp_tile_offset_n;
    C += C_offset_m * N + C_offset_n;

    // uncoalesced memory write
    // fixing it doesn't seem to make the kernel faster.
    // vectorized write is slower.
    for (int mma_m = 0; mma_m < WARP_M; mma_m += MMA_M)
        for (int mma_n = 0; mma_n < WARP_N; mma_n += MMA_N)
            for (int tm = 0; tm < THREAD_M; tm++)
                for (int tn = 0; tn < THREAD_N; tn++)
                    if ((C_offset_m + mma_m + tm < M) && (C_offset_n + mma_n + tn < N))
                        C[(mma_m + tm) * N + (mma_n + tn)] = acc[mma_m / MMA_M][mma_n / MMA_N][tm][tn];
}

void matmul_v5(const float *A, const float *B, float *C, int M, int N, int K) {
    // this config will result in identical kernel as v4

    const int BLOCK_M = 128, BLOCK_N = 64, BLOCK_K = 64;
    const int WARP_M = 32, WARP_N = 32;
    const int MMA_M = 16, MMA_N = 32;
    const int THREAD_N = 4;  // THREAD_M = MMA_M * MMA_N / 32 / THREAD_N = 4

    const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;  // 256
    const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
    matmul_v5_kernel<BLOCK_M, BLOCK_N, BLOCK_K, WARP_M, WARP_N, MMA_M, MMA_N, THREAD_N><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

// no bounds check
// NOTE: loop can be unrolled now since everything is known at compile time
// this gives a small boost. for load_shmem() (non-vectorized), this is actually slower.
template <int BLOCK_SIZE, int HEIGHT, int WIDTH, bool TRANSPOSED>
__device__ void load_shmem_vectorized(const float *in, int in_row_stride, float *out, int tid) {
  for (int offset = 0; offset < HEIGHT * WIDTH; offset += BLOCK_SIZE * 4) {
    const int idx = offset + tid * 4;
    const int row = idx / WIDTH;
    const int col = idx % WIDTH;

    float4 tmp = reinterpret_cast<const float4 *>(&in[row * in_row_stride + col])[0];

    if (TRANSPOSED) {
      out[(col + 0) * HEIGHT + row] = tmp.x;
      out[(col + 1) * HEIGHT + row] = tmp.y;
      out[(col + 2) * HEIGHT + row] = tmp.z;
      out[(col + 3) * HEIGHT + row] = tmp.w;
    } else
      reinterpret_cast<float4 *>(&out[row * WIDTH + col])[0] = tmp;
  }
}

// vectorized memory access without bounds check
// only memory access is different from v5
template <int BLOCK_M, int BLOCK_N, int BLOCK_K, int WARP_M, int WARP_N, int MMA_M, int MMA_N, int THREAD_N, bool TRANSPOSE_A_shmem>
__global__ void matmul_v6_kernel(const float *A, const float *B, float *C, int M, int N, int K) {
    static_assert(BLOCK_M % WARP_M == 0);
    static_assert(BLOCK_N % WARP_N == 0);
    static_assert(WARP_M % MMA_M == 0);
    static_assert(WARP_N % MMA_N == 0);
    static_assert((MMA_M * MMA_N / THREAD_N) % WARP_SIZE == 0);
    static_assert(THREAD_N % 4 == 0);  // so we can use vectorized access
    constexpr int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;
    constexpr int NUM_MMA_M = WARP_M / MMA_M;
    constexpr int NUM_MMA_N = WARP_N / MMA_N;
    constexpr int THREAD_M = MMA_M * MMA_N / (WARP_SIZE * THREAD_N);

    const int tid = threadIdx.x;
    const int block_id = blockIdx.x;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;

    const int num_blocks_per_row = cdiv(N, BLOCK_N);
    const int block_id_m = block_id / num_blocks_per_row;
    const int block_id_n = block_id % num_blocks_per_row;
    const int offset_m = block_id_m * BLOCK_M;
    const int offset_n = block_id_n * BLOCK_N;

    constexpr int num_warps_per_row = BLOCK_N / WARP_N;
    const int warp_id_m = warp_id / num_warps_per_row;
    const int warp_id_n = warp_id % num_warps_per_row;
    const int warp_tile_offset_m = warp_id_m * WARP_M;
    const int warp_tile_offset_n = warp_id_n * WARP_N;

    constexpr int num_thread_tiles_per_row = MMA_N / THREAD_N;
    const int thread_tile_id_m = lane_id / num_thread_tiles_per_row;
    const int thread_tile_id_n = lane_id % num_thread_tiles_per_row;
    const int thread_tile_offset_m = thread_tile_id_m * THREAD_M;
    const int thread_tile_offset_n = thread_tile_id_n * THREAD_N;

    A += offset_m * K;
    B += offset_n;

    __shared__ float A_shmem[BLOCK_M * BLOCK_K];
    __shared__ float B_shmem[BLOCK_K * BLOCK_N];
    float acc[NUM_MMA_M][NUM_MMA_N][THREAD_M][THREAD_N] = {0.0f};

    const float *A_thread_tile = reinterpret_cast<const float *>(A_shmem) + (warp_tile_offset_m + thread_tile_offset_m) * (TRANSPOSE_A_shmem ? 1 : BLOCK_K);
    const float *B_thread_tile = reinterpret_cast<const float *>(B_shmem) + (warp_tile_offset_n + thread_tile_offset_n);

    for (int offset_k = 0; offset_k < K; offset_k += BLOCK_K) {
        load_shmem_vectorized<BLOCK_SIZE, BLOCK_M, BLOCK_K, TRANSPOSE_A_shmem>(A, K, A_shmem, tid);
        load_shmem_vectorized<BLOCK_SIZE, BLOCK_K, BLOCK_N, false>(B, N, B_shmem, tid);
        __syncthreads();

        for (int k = 0; k < BLOCK_K; k++) {
            float A_reg[NUM_MMA_M][THREAD_M];
            float B_reg[NUM_MMA_N][THREAD_N];

            for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
                if (TRANSPOSE_A_shmem) {
                    static_assert(THREAD_M % 4 == 0);
                    for (int tm = 0; tm < THREAD_M; tm += 4) {
                        float4 tmp = reinterpret_cast<const float4 *>(&A_thread_tile[k * BLOCK_M + (mma_tile_id_m * MMA_M + tm)])[0];
                        reinterpret_cast<float4 *>(&A_reg[mma_tile_id_m][tm])[0] = tmp;
                    }
                }
                else {
                    for (int tm = 0; tm < THREAD_M; tm++)
                        A_reg[mma_tile_id_m][tm] = A_thread_tile[(mma_tile_id_m * MMA_M + tm) * BLOCK_K + k];
                }

            for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++)
                for (int tn = 0; tn < THREAD_N; tn += 4) {
                    float4 tmp = reinterpret_cast<const float4 *>(&B_thread_tile[k * BLOCK_N + (mma_tile_id_n * MMA_N + tn)])[0];
                    reinterpret_cast<float4 *>(&B_reg[mma_tile_id_n][tn])[0] = tmp;
                }

            for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
                for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++)
                    for (int tm = 0; tm < THREAD_M; tm++)
                        for (int tn = 0; tn < THREAD_N; tn++)
                            acc[mma_tile_id_m][mma_tile_id_n][tm][tn] += A_reg[mma_tile_id_m][tm] * B_reg[mma_tile_id_n][tn];
        }
        __syncthreads();

        A += BLOCK_K;
        B += BLOCK_K * N;
  }

    const int C_offset_m = offset_m + warp_tile_offset_m + thread_tile_offset_m;
    const int C_offset_n = offset_n + warp_tile_offset_n + thread_tile_offset_n;
    C += C_offset_m * N + C_offset_n;

    for (int mma_m = 0; mma_m < WARP_M; mma_m += MMA_M)
        for (int mma_n = 0; mma_n < WARP_N; mma_n += MMA_N)
            for (int tm = 0; tm < THREAD_M; tm++)
                for (int tn = 0; tn < THREAD_N; tn += 4) {
                    const float4 tmp = reinterpret_cast<const float4 *>(&acc[mma_m / MMA_M][mma_n / MMA_N][tm][tn])[0];
                    reinterpret_cast<float4 *>(&C[(mma_m + tm) * N + (mma_n + tn)])[0] = tmp;
                }
}

void matmul_v6a(const float *A, const float *B, float *C, int M, int N, int K) {
    assert(is_power_of_two(M) && "M must be a power of 2");
    assert(is_power_of_two(N) && "N must be a power of 2");
    assert(is_power_of_two(K) && "K must be a power of 2");

    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 16;
    const int WARP_M = 64, WARP_N = 64;
    const int MMA_M = 16, MMA_N = 32;
    const int THREAD_N = 4;  // THREAD_M = MMA_M * MMA_N / 32 / THREAD_N = 4

    const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;  // 128
    const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
    matmul_v6_kernel<BLOCK_M, BLOCK_N, BLOCK_K, WARP_M, WARP_N, MMA_M, MMA_N, THREAD_N, false><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

void matmul_v6b(const float *A, const float *B, float *C, int M, int N, int K) {
    assert(is_power_of_two(M) && "M must be a power of 2");
    assert(is_power_of_two(N) && "N must be a power of 2");
    assert(is_power_of_two(K) && "K must be a power of 2");

    const int BLOCK_M = 64, BLOCK_N = 128, BLOCK_K = 8;
    const int WARP_M = 64, WARP_N = 64;
    const int MMA_M = 32, MMA_N = 32;
    const int THREAD_N = 4;  // THREAD_M = MMA_M * MMA_N / 32 / THREAD_N = 8

    const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;  // 64
    const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
    matmul_v6_kernel<BLOCK_M, BLOCK_N, BLOCK_K, WARP_M, WARP_N, MMA_M, MMA_N, THREAD_N, true><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}