#include "hip/hip_runtime.h"
#include "mma.cuh"
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <cstdint>
#include <hip/hip_bf16.h>

#define PRINT_IF(cond, ...) if (cond) printf(__VA_ARGS__);

__host__ __device__ constexpr int cdiv(int a, int b) { return (a + b - 1) / b; }
constexpr bool is_power_of_two(int x) { return x > 0 && (x & (x - 1)) == 0; }  // https://stackoverflow.com/a/1804686
constexpr int WARP_SIZE = 32;

template <int BLOCK_SIZE, int HEIGHT, int WIDTH, typename T>
__device__ void load_b128(const T *in, int in_row_stride, T *out, int out_row_stride, int tid) {
    // number of elements to do 128-bit/16-byte load
    // e.g. FP32 -> 4 elements, BF16 -> 8 elements.
    using load_type = uint4;
    constexpr int num_elems = sizeof(load_type) / sizeof(T);

    for (int idx = tid * num_elems; idx < HEIGHT * WIDTH; idx += BLOCK_SIZE * num_elems) {
        const int row = idx / WIDTH;
        const int col = idx % WIDTH;
        load_type tmp = reinterpret_cast<const load_type *>(&in[row * in_row_stride + col])[0];
        reinterpret_cast<load_type *>(&out[row * out_row_stride + col])[0] = tmp;
    }
}

template <typename T> __device__ ushort f32_to_b16(float x);
template <> __device__ ushort f32_to_b16<half>(float x) { return __half_as_ushort(__float2half(x)); }
template <> __device__ ushort f32_to_b16<hip_bfloat16>(float x) { return __bfloat16_as_ushort(__float2bfloat16(x)); }

template <
  int BLOCK_M, int BLOCK_N, int BLOCK_K,
  int WARP_M, int WARP_N, int WARP_K,
  int MMA_M, int MMA_N, int MMA_K,
  bool PAD_SHMEM_A, bool PAD_SHMEM_B,
  typename T>
__global__ void matmul_v1_kernel(const T *A, const T *B, T *C, int M, int N, int K) {
    static_assert(BLOCK_M % WARP_M == 0);
    static_assert(BLOCK_N % WARP_N == 0);
    static_assert(BLOCK_K % WARP_K == 0);
    static_assert(WARP_M % MMA_M == 0);
    static_assert(WARP_N % MMA_N == 0);
    static_assert(WARP_K % MMA_K == 0);
    constexpr int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;
    constexpr int NUM_MMA_M = WARP_M / MMA_M;
    constexpr int NUM_MMA_N = WARP_N / MMA_N;
    constexpr int NUM_MMA_K = WARP_K / MMA_K;

    const int tid = threadIdx.x;
    const int block_id = blockIdx.x;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;

    const int num_blocks_per_row = cdiv(N, BLOCK_N);
    const int block_id_m = block_id / num_blocks_per_row;
    const int block_id_n = block_id % num_blocks_per_row;
    const int offset_m = block_id_m * BLOCK_M;
    const int offset_n = block_id_n * BLOCK_N;

    constexpr int num_warps_per_row = BLOCK_N / WARP_N;
    const int warp_id_m = warp_id / num_warps_per_row;
    const int warp_id_n = warp_id % num_warps_per_row;
    const int warp_tile_offset_m = warp_id_m * WARP_M;
    const int warp_tile_offset_n = warp_id_n * WARP_N;

    // A is row-major, B is column-major
    A += offset_m * K;
    B += offset_n * K;

    // we can only pad 8 elements = 16 bytes to ensure 16-byte alignment required by ldmatrix
    constexpr int A_shared_width = BLOCK_K + (PAD_SHMEM_A ? 8 : 0);
    constexpr int B_shared_width = BLOCK_K + (PAD_SHMEM_B ? 8 : 0);
    __shared__ T A_shared[BLOCK_M * A_shared_width];
    __shared__ T B_shared[BLOCK_N * B_shared_width];

    // 32-bit (4-byte) registers
    constexpr int num_acc_regs = MMA_M * MMA_N / WARP_SIZE;
    constexpr int num_A_regs = MMA_M * MMA_K * sizeof(T) / 4 / WARP_SIZE;
    constexpr int num_B_regs = MMA_N * MMA_K * sizeof(T) / 4 / WARP_SIZE;
    float acc[NUM_MMA_M][NUM_MMA_N][num_acc_regs] = {0.0f};  // for m16n8k8, each thread holds 4 output float
    uint32_t A_reg[NUM_MMA_M][NUM_MMA_K][num_A_regs];        //              each thread holds 2 input f16x2
    uint32_t B_reg[NUM_MMA_N][NUM_MMA_K][num_B_regs];        //              each thread holds 1 input f16x1

    // first A and B warp-tile along BLOCK_K dim (we will iterate along BLOCK_K with step_size=WARP_K)
    const T *A_warp_tile = reinterpret_cast<const T *>(A_shared) + warp_tile_offset_m * A_shared_width;
    const T *B_warp_tile = reinterpret_cast<const T *>(B_shared) + warp_tile_offset_n * B_shared_width;

    for (int block_k = 0; block_k < K; block_k += BLOCK_K) {
        load_b128<BLOCK_SIZE, BLOCK_M, BLOCK_K>(A, K, A_shared, A_shared_width, tid);
        load_b128<BLOCK_SIZE, BLOCK_N, BLOCK_K>(B, K, B_shared, B_shared_width, tid);
        __syncthreads();

        for (int warp_k = 0; warp_k < BLOCK_K; warp_k += WARP_K) {
            // load data from shared memory to registers using ldmatrix
            // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-instructions-ldmatrix

            // convert generic address to .shared state space address expected by inline PTX
            // thread 0 holds address of row 0
            // thread 1 holds address of row 1, and so on
            uint32_t A_tile_addr = cvta_shared(A_warp_tile + lane_id * A_shared_width + warp_k);
            uint32_t B_tile_addr = cvta_shared(B_warp_tile + lane_id * B_shared_width + warp_k);

            // load A to registers
            // ldmatrix can only load 8x8 matrix. for 16x8 tile, we need to use x2
            // works for both m16n8k8 and m16n8k16
            for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
                for (int mma_tile_id_k = 0; mma_tile_id_k < NUM_MMA_K; mma_tile_id_k++) {
                uint32_t A_local = A_tile_addr + (mma_tile_id_m * MMA_M * A_shared_width + mma_tile_id_k * MMA_K) * sizeof(T);
                ldmatrix<num_A_regs>(A_reg[mma_tile_id_m][mma_tile_id_k], A_local);
                }

            // load B to registers
            for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++)
                for (int mma_tile_id_k = 0; mma_tile_id_k < NUM_MMA_K; mma_tile_id_k++) {
                uint32_t B_local = B_tile_addr + (mma_tile_id_n * MMA_N * B_shared_width + mma_tile_id_k * MMA_K) * sizeof(T);
                ldmatrix<num_B_regs>(B_reg[mma_tile_id_n][mma_tile_id_k], B_local);
                }

            // call mma
            // https://docs.nvidia.com/cuda/parallel-thread-execution/#warp-level-matrix-fragment-mma-1688
            for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
                for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++)
                for (int mma_tile_id_k = 0; mma_tile_id_k < NUM_MMA_K; mma_tile_id_k++)
                    mma<MMA_M, MMA_N, MMA_K, T>(A_reg[mma_tile_id_m][mma_tile_id_k],
                                                B_reg[mma_tile_id_n][mma_tile_id_k],
                                                acc[mma_tile_id_m][mma_tile_id_n]);
        }
        __syncthreads();

        A += BLOCK_K;
        B += BLOCK_K;
    }

    const int C_offset_m = offset_m + warp_tile_offset_m;
    const int C_offset_n = offset_n + warp_tile_offset_n;
    C += C_offset_m * N + C_offset_n;

    // check output layout here
    // https://docs.nvidia.com/cuda/parallel-thread-execution/#mma-1688-c-f16-f32
    // m16n8k16 has the same layout
    const int a0_row = lane_id >> 2;
    const int a0_col = (lane_id % 4) * 2;
    C += a0_row * N + a0_col;

    for (int mma_tile_id_m = 0; mma_tile_id_m < NUM_MMA_M; mma_tile_id_m++)
        for (int mma_tile_id_n = 0; mma_tile_id_n < NUM_MMA_N; mma_tile_id_n++) {
        T *C_local = C + mma_tile_id_m * MMA_M * N + mma_tile_id_n * MMA_N;
        float *acc_frag = acc[mma_tile_id_m][mma_tile_id_n];
        ushort2 tmp;

        // write a0 and a1
        tmp.x = f32_to_b16<T>(acc_frag[0]);
        tmp.y = f32_to_b16<T>(acc_frag[1]);
        reinterpret_cast<ushort2 *>(C_local)[0] = tmp;

        // write a2 and a3
        tmp.x = f32_to_b16<T>(acc_frag[2]);
        tmp.y = f32_to_b16<T>(acc_frag[3]);
        reinterpret_cast<ushort2 *>(C_local + 8 * N)[0] = tmp;
        }
}

void matmul_v1a(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
    assert(is_power_of_two(M) && "M must be a power of 2");
    assert(is_power_of_two(N) && "N must be a power of 2");
    assert(is_power_of_two(K) && "K must be a power of 2");

    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
    const int WARP_M = 64, WARP_N = 64, WARP_K = 16;
    const int MMA_M = 16, MMA_N = 8, MMA_K = 8;

    const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;
    const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
    matmul_v1_kernel<
        BLOCK_M, BLOCK_N, BLOCK_K,
        WARP_M, WARP_N, WARP_K,
        MMA_M, MMA_N, MMA_K,
        false, false><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}

void matmul_v1b(const hip_bfloat16 *A, const hip_bfloat16 *B, hip_bfloat16 *C, int M, int N, int K) {
    assert(is_power_of_two(M) && "M must be a power of 2");
    assert(is_power_of_two(N) && "N must be a power of 2");
    assert(is_power_of_two(K) && "K must be a power of 2");

    const int BLOCK_M = 128, BLOCK_N = 128, BLOCK_K = 32;
    const int WARP_M = 64, WARP_N = 64, WARP_K = 16;
    const int MMA_M = 16, MMA_N = 8, MMA_K = 8;

    const int BLOCK_SIZE = (BLOCK_M * BLOCK_N) / (WARP_M * WARP_N) * WARP_SIZE;
    const int grid_size = cdiv(M * N, BLOCK_M * BLOCK_N);
    matmul_v1_kernel<
        BLOCK_M, BLOCK_N, BLOCK_K,
        WARP_M, WARP_N, WARP_K,
        MMA_M, MMA_N, MMA_K,
        true, false><<<grid_size, BLOCK_SIZE>>>(A, B, C, M, N, K);
}
